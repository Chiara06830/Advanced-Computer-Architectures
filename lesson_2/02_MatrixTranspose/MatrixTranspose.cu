#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

const int N  = 1024;
const int BLOCK_SIZE = 16;

__global__
void matrixTransposeKernel(const int* d_matrix_in,
                           int        N,
                           int*       d_matrix_out) {
    __shared__ int ds[BLOCK_SIZE][BLOCK_SIZE];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;         
    int col = bx * BLOCK_SIZE + tx;
	int row = by * BLOCK_SIZE + ty;
    
    ds[tx][ty] = d_matrix_in[col * N + row];
    __syncthreads();
    
    d_matrix_out[row * N + col] = ds[tx][ty];
    
}

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrix_in  = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrix_out = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++)
        h_matrix_in[i] = distribution(generator);
    

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    std::cout<<"Starting computation on HOST.."<<std::endl;
    TM_host.start();

    for (int i = 0; i < N ; i++) {
        for (int j = 0; j < N ; j++)
            h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
    }

    TM_host.stop();
    TM_host.print("MatrixTranspose host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrix_in, *d_matrix_out;
    SAFE_CALL( hipMalloc( &d_matrix_in, N*N * sizeof(int) ));
    SAFE_CALL( hipMalloc( &d_matrix_out, N*N * sizeof(int) ));
    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrix_in, h_matrix_in, N*N * sizeof(int), hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------
    // DEVICE INIT
    dim3 DimGrid(N/BLOCK_SIZE, N/BLOCK_SIZE, 1);
    if (N%BLOCK_SIZE) { DimGrid.x++; DimGrid.y++; }
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();

    matrixTransposeKernel<<< DimGrid,DimBlock>>> (d_matrix_in, N, d_matrix_out);

    CHECK_CUDA_ERROR
    TM_device.stop();
    
    TM_device.print("MatrixTranspose device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrix_out, N*N * sizeof(int), hipMemcpyDeviceToHost));

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N * N; i++) {
        if (h_matrix_out[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrix_out[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix_in;
    delete[] h_matrix_out;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrix_in ) );
    SAFE_CALL( hipFree( d_matrix_out ) );

    // -------------------------------------------------------------------------
    //hipDeviceReset();
}
